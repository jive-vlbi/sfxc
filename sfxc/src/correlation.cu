
#include <hip/hip_runtime.h>
__global__ void __autocorr(float2 *s, float2 *d, int n, int stride)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	
	for (int j = i; j < n * stride; j += stride) {
	        d[i].x += s[j].x * s[j].x + s[j].y * s[j].y;
	        d[i].y += s[j].y * s[j].x - s[j].x * s[j].y;
	}
}

extern "C" void autocorr(float2 *s, float2 *d, int n, int stride)
{
	__autocorr<<<stride / 256, 256>>>(s, d, n, stride);
}

__global__ void __crosscorr(float2 *s1, float2 *s2, float2 *d, int n, int stride)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	
	for (int j = i; j < n * stride; j += stride) {
	        d[i].x += s1[j].x * s2[j].x + s1[j].y * s2[j].y;
	        d[i].y += s1[j].y * s2[j].x - s1[j].x * s2[j].y;
	}
}

extern "C" void crosscorr(float2 *s1, float2 *s2, float2 *d, int n, int stride)
{
	__crosscorr<<<stride / 256, 256>>>(s1, s2, d, n, stride);
}
